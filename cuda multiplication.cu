#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ matrixMultiply(const float *matA, const float *matB, const float *r, int matrixSize){
    int indexColumn = threadIdx.x + blockDimx.x * blockIdx.x;
    int indexRow = threadIdx.y + blockDimx.y + blockIdx.y;

}

__global__ expressionCalculator(const float *A, const float *B, const float *C, const float *result, int matrixSize){
}

int main(){
    float *hostA, *hostB, *hostC, *hostResult, *deviceA, *deviceB, *deviceC, *deviceResult;

    // Allocate memory in host memory space
    int elements = MATRIX_SIZE * MATRIX_SIZE;
    hostA = new float[elements];
    hostB = new float[elements];
    hostC = new float[elements];
    hostResult = new float[elements];
    
    for (int i=0; i<elements; i++){
        hostA[i] = rand();
        hostB[i] = rand();
        hostC[i] = rand();
    }
    
     // Allocate in device memory space
    hipMalloc(&deviceA, elements*sizeof(float));
    hipMalloc(&deviceB, elements*sizeof(float));
    hipMalloc(&deviceC, elements*sizeof(float));
    hipMalloc(&deviceResult, elements*sizeof(float));

    hipMemcpy(deviceA, hostA, elements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, elements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, hostC, elements*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Finished execution\n");

    return 0;
}
